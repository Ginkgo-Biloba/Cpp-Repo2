﻿
#include <hip/hip_runtime.h>
#include <>
#include "../common/base.hpp"


int main()
{
	int count;
	CheckError(hipGetDeviceCount(&count));
	printf("cuda count: %d\n", count);

	hipDeviceProp_t prop;
	CheckError(hipGetDeviceProperties(&prop, 0));
	printf("cuda core number per sp: %d\n", getCorePerSP(prop));

	int driver;
	CheckError(hipDriverGetVersion(&driver));
	printf("cuda driver version: %d.%d\n",
		driver / 1000, (driver % 1000) / 10);

	int runtime;
	CheckError(hipRuntimeGetVersion(&runtime));
	printf("cuda rumtime version: %d.%d\n",
		runtime / 1000, (runtime % 1000) / 10);

	puts("hello cuda");
}
